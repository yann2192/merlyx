#include <stdio.h>

void printProperties(struct hipDeviceProp_t * p){
	printf("Device name : %s \n", p->name);
	printf("Version : %d.%d\n",p->major, p->minor);
	printf("Global memory available (in byte) : %d\n", p->totalGlobalMem);
	printf("Shared memory available (by thread in byte) : %d\n", p->sharedMemPerBlock);
	printf("32-bit registers available (by thread) : %d\n", p->regsPerBlock);
	printf("The warp size in threads : %d\n", p->warpSize);
	printf("Maximum threads per blocks : %d\n", p->maxThreadsPerBlock);
	printf("Clock rate (in kHz) : %d\n", p->clockRate);
	printf("Total const memory (in byte) : %d\n", p->totalConstMem);
	printf("Texture alignment (in byte) : %d\n", p->textureAlignment);
	printf("Number of multiprocessors : %d\n", p->multiProcessorCount);
	printf("Run time limit? : %d\n", p->kernelExecTimeoutEnabled);
	printf("memPitch (in byte): %d\n", p->memPitch);
	printf("Maximum size of each dimension of a block : %d %d %d\n", p->maxThreadsDim[0], p->maxThreadsDim[1],p->maxThreadsDim[2]);
	printf("maximum size of each dimension of a grid : %d %d %d\n", p->maxGridSize[0], p->maxGridSize[1],p->maxGridSize[2]);
	printf("deviceOverlap : %d\n", p->deviceOverlap);
	printf("integrated : %\n", p->integrated);
	printf("canMapHostMemory : %\n", p->canMapHostMemory);
	printf("computeMode : %s\n", p->computeMode);
} 

int main(){
	struct hipDeviceProp_t deviceProp;
	int numDevice;
	
	if(hipChooseDevice(&numDevice, &deviceProp) == hipSuccess){
		if(cudaDeviceProperties(&deviceProp, numDevice) == hipErrorInvalidDevice)
			printf("Error, selected device not found");
		else
			printProperties(&deviceProp);
	}else{
		printf("Error with hipChooseDevice");
	}
	return 0;
}